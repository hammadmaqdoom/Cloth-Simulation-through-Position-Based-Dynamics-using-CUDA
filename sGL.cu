#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Movania Muhammad Mobeen
All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list
of conditions and the following disclaimer.
Redistributions in binary form must reproduce the above copyright notice, this list
of conditions and the following disclaimer in the documentation and/or other
materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.
*/

//A simple cloth using position based dynamics based on the SIGGRAPH course notes
//"Realtime Physics" http://www.matthiasmueller.info/realtimephysics/coursenotes.pdf using 
//GLUT,GLEW and GLM libraries. This code is intended for beginners so that they may 
//understand what is required to implement position based dynamics based cloth simulation.
//
//This code is under BSD license. If you make some improvements,
//or are using this in your research, do let me know and I would appreciate
//if you acknowledge this in your code or in your publication.
//
//Controls:
//left click on any empty region to rotate, middle click to zoom 
//left click and drag any point to drag it.
//
//Author: Movania Muhammad Mobeen
//        School of Computer Engineering,
//        Nanyang Technological University,
//        Singapore.
//Email : mova0002@e.ntu.edu.sg 
//

#include <GL/glew.h>
#include <GL/wglew.h>
#include <GL/freeglut.h>
#include <vector>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp> //for matrices
#include <glm/gtc/type_ptr.hpp>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <hip/hip_vector_types.h>

//undefine if u want to use the default bending constraint of pbd
#define USE_TRIANGLE_BENDING_CONSTRAINT

#pragma comment(lib, "glew32.lib")

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource* cuda_vbo_resource;
void* d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface* timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int* pArgc = NULL;
char** pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char** argv, char* ref_file);
void cleanup();

// GL functionality
bool initGL(int* argc, char** argv);
void createVBO(GLuint* vbo, struct hipGraphicsResource** vbo_res,
	unsigned int vbo_res_flags);
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource** vbo_resource);
void runAutoTest(int devID, char** argv, char* ref_file);
void checkResultCuda(int argc, char** argv, const GLuint& vbo);

const char* sSDKsample = "simpleGL (VBO)";

using namespace std;
const int width = 1024, height = 1024;

#define PI 3.1415926536f
#define EPSILON  0.0000001f

int numX = 20, numY = 20; //these ar the number of quads
const size_t total_points = (numX + 1) * (numY + 1);
float fullsize = 4.0f;
float halfsize = fullsize / 2.0f;

char info[MAX_PATH] = { 0 };

float timeStep = 1.0f / 60.0f; //1.0/60.0f;
float currentTime = 0;
double accumulator = timeStep;
int selected_index = -1;
float global_dampening = 0.98f; //DevO: 24.07.2011  //global velocity dampening !!!

struct DistanceConstraint { int p1, p2;	float rest_length, k; float k_prime; };
#ifdef USE_TRIANGLE_BENDING_CONSTRAINT
struct BendingConstraint { int p1, p2, p3;	float rest_length, w, k; float k_prime; };
#else
struct BendingConstraint { int p1, p2, p3, p4;	float rest_length1, rest_length2, w1, w2, k; float k_prime; };
#endif

vector<GLushort> indices;
vector<DistanceConstraint> d_constraints;

vector<BendingConstraint> b_constraints;
vector<float> phi0; //initial dihedral angle between adjacent triangles

//particle system
vector<glm::vec3> X; //position
vector<glm::vec3> tmp_X; //predicted position
vector<glm::vec3> V; //velocity
vector<glm::vec3> F;
vector<float> W; //inverse particle mass 
vector<glm::vec3> Ri; //Ri = Xi-Xcm 

int oldX = 0, oldY = 0;
float rX = 15, rY = 0;
int state = 1;
float dist = -23;
const int GRID_SIZE = 10;

const size_t solver_iterations = 2; //number of solver iterations per step. PBD  

float kBend = 0.5f;
float kStretch = 0.25f;
float kDamp = 0.00125f;
glm::vec3 gravity = glm::vec3(0.0f, -0.00981f, 0.0f);

float mass = 1.f / (total_points);


GLint viewport[4];
GLdouble MV[16];
GLdouble P[16];

LARGE_INTEGER frequency;        // ticks per second
LARGE_INTEGER t1, t2;           // ticks
double frameTimeQP = 0;
float frameTime = 0;


glm::vec3 Up = glm::vec3(0, 1, 0), Right, viewDir;
float startTime = 0, fps = 0;
int totalFrames = 0;

__global__ void kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	// calculate coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	float freq = 4.0f;
	float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

	// write output vertex
	pos[y * width + x] = make_float4(u, w, v, 1.0f);
}


void launch_kernel(float4* pos, unsigned int mesh_width,
	unsigned int mesh_height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}


__device__ void StepPhysics(float dt);

__device__
float GetArea(int a, int b, int c) {
	glm::vec3 e1 = X[b] - X[a];
	glm::vec3 e2 = X[c] - X[a];
	return 0.5f * glm::length(glm::cross(e1, e2));
}
__device__ 
void AddDistanceConstraint(int a, int b, float k) {
	DistanceConstraint c;
	c.p1 = a;
	c.p2 = b;
	c.k = k;
	c.k_prime = 1.0f - pow((1.0f - c.k), 1.0f / solver_iterations);  //1.0f-pow((1.0f-c.k), 1.0f/ns);

	if (c.k_prime > 1.0)
		c.k_prime = 1.0;

	glm::vec3 deltaP = X[c.p1] - X[c.p2];
	c.rest_length = glm::length(deltaP);

	d_constraints.push_back(c);
}
#ifdef USE_TRIANGLE_BENDING_CONSTRAINT
__device__ 
void AddBendingConstraint(int pa, int pb, int pc, float k) {
	BendingConstraint c;
	c.p1 = pa;
	c.p2 = pb;
	c.p3 = pc;

	c.w = W[pa] + W[pb] + 2 * W[pc];
	glm::vec3 center = 0.3333f * (X[pa] + X[pb] + X[pc]);
	c.rest_length = glm::length(X[pc] - center);
	c.k = k;
	c.k_prime = 1.0f - pow((1.0f - c.k), 1.0f / solver_iterations);  //1.0f-pow((1.0f-c.k), 1.0f/ns);
	if (c.k_prime > 1.0)
		c.k_prime = 1.0;
	b_constraints.push_back(c);
}
#else
void AddBendingConstraint(int pa, int pb, int pc, int pd, float k) {
	BendingConstraint c;
	c.p1 = pa;
	c.p2 = pb;
	c.p3 = pc;
	c.p4 = pd;
	c.w1 = W[pa] + W[pb] + 2 * W[pc];
	c.w2 = W[pa] + W[pb] + 2 * W[pd];
	glm::vec3 center1 = 0.3333f * (X[pa] + X[pb] + X[pc]);
	glm::vec3 center2 = 0.3333f * (X[pa] + X[pb] + X[pd]);
	c.rest_length1 = glm::length(X[pc] - center1);
	c.rest_length2 = glm::length(X[pd] - center2);
	c.k = k;

	c.k_prime = 1.0f - pow((1.0f - c.k), 1.0f / solver_iterations);  //1.0f-pow((1.0f-c.k), 1.0f/ns);
	if (c.k_prime > 1.0)
		c.k_prime = 1.0;
	b_constraints.push_back(c);
}
#endif
void OnMouseDown(int button, int s, int x, int y)
{
	if (s == GLUT_DOWN)
	{
		oldX = x;
		oldY = y;
		int window_y = (height - y);
		float norm_y = float(window_y) / float(height / 2.0);
		int window_x = x;
		float norm_x = float(window_x) / float(width / 2.0);

		float winZ = 0;
		glReadPixels(x, height - y, 1, 1, GL_DEPTH_COMPONENT, GL_FLOAT, &winZ);
		if (winZ == 1)
			winZ = 0;
		double objX = 0, objY = 0, objZ = 0;
		gluUnProject(window_x, window_y, winZ, MV, P, viewport, &objX, &objY, &objZ);
		glm::vec3 pt(objX, objY, objZ);
		size_t i = 0;
		for (i = 0; i < total_points; i++) {
			if (glm::distance(X[i], pt) < 0.1) {
				selected_index = i;
				printf("Intersected at %d\n", i);
				break;
			}
		}
	}

	if (button == GLUT_MIDDLE_BUTTON)
		state = 0;
	else
		state = 1;

	if (s == GLUT_UP) {
		selected_index = -1;
		glutSetCursor(GLUT_CURSOR_INHERIT);
	}
}

void OnMouseMove(int x, int y)
{
	if (selected_index == -1) {
		if (state == 0)
			dist *= (1 + (y - oldY) / 60.0f);
		else
		{
			rY += (x - oldX) / 5.0f;
			rX += (y - oldY) / 5.0f;
		}
	}
	else {
		float delta = 1500 / abs(dist);
		float valX = (x - oldX) / delta;
		float valY = (oldY - y) / delta;
		if (abs(valX) > abs(valY))
			glutSetCursor(GLUT_CURSOR_LEFT_RIGHT);
		else
			glutSetCursor(GLUT_CURSOR_UP_DOWN);



		V[selected_index] = glm::vec3(0);
		//X[selected_index].x += Right[0]*valX ;
		//float newValue = X[selected_index].y+Up[1]*valY;
		//if(newValue>0)
		//	X[selected_index].y = newValue;
		//X[selected_index].z += Right[2]*valX + Up[2]*valY;
		X[selected_index].x += Right[0] * valX + Up[0] * valY;
		X[selected_index].y += Right[1] * valX + Up[1] * valY;
		X[selected_index].z += Right[2] * valX + Up[2] * valY;
	}
	oldX = x;
	oldY = y;

	glutPostRedisplay();
}


__device__
void DrawGrid()
{
	glBegin(GL_LINES);
	glColor3f(0.5f, 0.5f, 0.5f);
	for (int i = -GRID_SIZE; i <= GRID_SIZE; i++)
	{
		glVertex3f((float)i, 0, (float)-GRID_SIZE);
		glVertex3f((float)i, 0, (float)GRID_SIZE);

		glVertex3f((float)-GRID_SIZE, 0, (float)i);
		glVertex3f((float)GRID_SIZE, 0, (float)i);
	}
	glEnd();
}

inline glm::vec3 GetNormal(int ind0, int ind1, int ind2) {
	glm::vec3 e1 = X[ind0] - X[ind1];
	glm::vec3 e2 = X[ind2] - X[ind1];
	return glm::normalize(glm::cross(e1, e2));
}

#ifndef USE_TRIANGLE_BENDING_CONSTRAINT
inline float GetDihedralAngle(BendingConstraint c, float& d, glm::vec3& n1, glm::vec3& n2) {
	n1 = GetNormal(c.p1, c.p2, c.p3);
	n2 = GetNormal(c.p1, c.p2, c.p4);
	d = glm::dot(n1, n2);
	return acos(d);
}
#else
inline int getIndex(int i, int j) {
	return j * (numX + 1) + i;
}
#endif
void InitGL() {

	startTime = (float)glutGet(GLUT_ELAPSED_TIME);
	currentTime = startTime;

	// get ticks per second
	QueryPerformanceFrequency(&frequency);

	// start timer
	QueryPerformanceCounter(&t1);


	glEnable(GL_DEPTH_TEST);
	size_t i = 0, j = 0, count = 0;
	int l1 = 0, l2 = 0;
	float ypos = 7.0f;
	int v = numY + 1;
	int u = numX + 1;

	indices.resize(numX * numY * 2 * 3);

	X.resize(total_points);
	tmp_X.resize(total_points);
	V.resize(total_points);
	F.resize(total_points);
	Ri.resize(total_points);

	//fill in positions
	for (int j = 0; j <= numY; j++) {
		for (int i = 0; i <= numX; i++) {
			X[count++] = glm::vec3(((float(i) / (u - 1)) * 2 - 1) * halfsize, fullsize + 1, ((float(j) / (v - 1)) * fullsize));
		}
	}

	///DevO: 24.07.2011
	W.resize(total_points);
	for (i = 0; i < total_points; i++) {
		W[i] = 1.0f / mass;
	}
	/// 2 Fixed Points 
	W[0] = 0.0;
	W[numX] = 0.0;

	memcpy(&tmp_X[0].x, &X[0].x, sizeof(glm::vec3) * total_points);

	//fill in velocities	 
	memset(&(V[0].x), 0, total_points * sizeof(glm::vec3));

	//fill in indices
	GLushort* id = &indices[0];
	for (int i = 0; i < numY; i++) {
		for (int j = 0; j < numX; j++) {
			int i0 = i * (numX + 1) + j;
			int i1 = i0 + 1;
			int i2 = i0 + (numX + 1);
			int i3 = i2 + 1;
			if ((j + i) % 2) {
				*id++ = i0; *id++ = i2; *id++ = i1;
				*id++ = i1; *id++ = i2; *id++ = i3;
			}
			else {
				*id++ = i0; *id++ = i2; *id++ = i3;
				*id++ = i0; *id++ = i3; *id++ = i1;
			}
		}
	}

	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	//glPolygonMode(GL_BACK, GL_LINE);
	glPointSize(5);

	wglSwapIntervalEXT(0);

	//check the damping values
	if (kStretch > 1)
		kStretch = 1;
	if (kStretch < 0)
		kStretch = 0;
	if (kBend > 1)
		kBend = 1;
	if (kBend < 0)
		kBend = 0;
	if (kDamp > 1)
		kDamp = 1;
	if (kDamp < 0)
		kDamp = 0;
	if (global_dampening > 1)
		global_dampening = 1;

	//setup constraints
	// Horizontal
	for (l1 = 0; l1 < v; l1++)	// v
		for (l2 = 0; l2 < (u - 1); l2++) {
			AddDistanceConstraint((l1 * u) + l2, (l1 * u) + l2 + 1, kStretch);
		}

	// Vertical
	for (l1 = 0; l1 < (u); l1++)
		for (l2 = 0; l2 < (v - 1); l2++) {
			AddDistanceConstraint((l2 * u) + l1, ((l2 + 1) * u) + l1, kStretch);
		}


	// Shearing distance constraint
	for (l1 = 0; l1 < (v - 1); l1++)
		for (l2 = 0; l2 < (u - 1); l2++) {
			AddDistanceConstraint((l1 * u) + l2, ((l1 + 1) * u) + l2 + 1, kStretch);
			AddDistanceConstraint(((l1 + 1) * u) + l2, (l1 * u) + l2 + 1, kStretch);
		}


	// create bending constraints	
#ifdef USE_TRIANGLE_BENDING_CONSTRAINT
//add vertical constraints
	for (int i = 0; i <= numX; i++) {
		for (int j = 0; j < numY - 1; j++) {
			AddBendingConstraint(getIndex(i, j), getIndex(i, (j + 1)), getIndex(i, j + 2), kBend);
		}
	}
	//add horizontal constraints
	for (int i = 0; i < numX - 1; i++) {
		for (int j = 0; j <= numY; j++) {
			AddBendingConstraint(getIndex(i, j), getIndex(i + 1, j), getIndex(i + 2, j), kBend);
		}
	}

#else
	for (int i = 0; i < v - 1; ++i) {
		for (int j = 0; j < u - 1; ++j) {
			int p1 = i * (numX + 1) + j;
			int p2 = p1 + 1;
			int p3 = p1 + (numX + 1);
			int p4 = p3 + 1;

			if ((j + i) % 2) {
				AddBendingConstraint(p3, p2, p1, p4, kBend);
			}
			else {
				AddBendingConstraint(p4, p1, p3, p2, kBend);
			}
		}
	}
	float d;
	glm::vec3 n1, n2;
	phi0.resize(b_constraints.size());

	for (i = 0; i < b_constraints.size(); i++) {
		phi0[i] = GetDihedralAngle(b_constraints[i], d, n1, n2);
	}
#endif


}

void OnReshape(int nw, int nh) {
	glViewport(0, 0, nw, nh);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60, (GLfloat)nw / (GLfloat)nh, 1.f, 100.0f);

	glGetIntegerv(GL_VIEWPORT, viewport);
	glGetDoublev(GL_PROJECTION_MATRIX, P);

	glMatrixMode(GL_MODELVIEW);
}

void OnRender() {
	size_t i = 0;
	float newTime = (float)glutGet(GLUT_ELAPSED_TIME);
	frameTime = newTime - currentTime;
	currentTime = newTime;
	//accumulator += frameTime;

	//Using high res. counter
	QueryPerformanceCounter(&t2);
	// compute and print the elapsed time in millisec
	frameTimeQP = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart;
	t1 = t2;
	accumulator += frameTimeQP;

	++totalFrames;
	if ((newTime - startTime) > 1000)
	{
		float elapsedTime = (newTime - startTime);
		fps = (totalFrames / elapsedTime) * 1000;
		startTime = newTime;
		totalFrames = 0;
	}

	sprintf_s(info, "FPS: %3.2f, Frame time (GLUT): %3.4f msecs, Frame time (QP): %3.3f", fps, frameTime, frameTimeQP);
	glutSetWindowTitle(info);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLoadIdentity();

	//set viewing transformation
	glTranslatef(0, 0, dist);
	glRotatef(rX, 1, 0, 0);
	glRotatef(rY, 0, 1, 0);

	glGetDoublev(GL_MODELVIEW_MATRIX, MV);
	viewDir.x = (float)-MV[2];
	viewDir.y = (float)-MV[6];
	viewDir.z = (float)-MV[10];
	//Right = glm::cross(viewDir, Up);
	Right.x = (float)MV[0];
	Right.y = (float)MV[4];
	Right.z = (float)MV[8];

	Up.x = (float)MV[1];
	Up.y = (float)MV[5];
	Up.z = (float)MV[9];

	//draw grid
	DrawGrid();




	//draw polygons
	glColor3f(1, 1, 1);
	glBegin(GL_TRIANGLES);
	for (i = 0; i < indices.size(); i += 3) {
		glm::vec3 p1 = X[indices[i]];
		glm::vec3 p2 = X[indices[i + 1]];
		glm::vec3 p3 = X[indices[i + 2]];
		glVertex3f(p1.x, p1.y, p1.z);
		glVertex3f(p2.x, p2.y, p2.z);
		glVertex3f(p3.x, p3.y, p3.z);
	}
	glEnd();

	//draw points

	glBegin(GL_POINTS);
	for (i = 0; i < total_points; i++) {
		glm::vec3 p = X[i];
		int is = (i == selected_index);
		glColor3f((float)!is, (float)is, (float)is);
		glVertex3f(p.x, p.y, p.z);
	}
	glEnd();


	//draw normals for debug only 	
#ifndef USE_TRIANGLE_BENDING_CONSTRAINT
#ifdef _DEBUG
	BendingConstraint b;
	float size = 0.1f;
	float d = 0;
	glm::vec3 n1, n2, c1, c2;


	glBegin(GL_LINES);
	for (i = 0; i < b_constraints.size(); i++) {
		b = b_constraints[i];
		c1 = (X[b.p1] + X[b.p2] + X[b.p3]) / 3.0f;
		c2 = (X[b.p1] + X[b.p2] + X[b.p4]) / 3.0f;
		GetDihedralAngle(b, d, n1, n2);
		glColor3f(abs(n1.x), abs(n1.y), abs(n1.z));
		glVertex3f(c1.x, c1.y, c1.z);		glVertex3f(c1.x + size * n1.x, c1.y + size * n1.y, c1.z + size * n1.z);

		glColor3f(abs(n2.x), abs(n2.y), abs(n2.z));
		glVertex3f(c2.x, c2.y, c2.z);		glVertex3f(c2.x + size * n2.x, c2.y + size * n2.y, c2.z + size * n2.z);
	}
	glEnd();
#endif
#endif
	glutSwapBuffers();
}

void OnShutdown() {
	d_constraints.clear();
	b_constraints.clear();
	indices.clear();
	X.clear();
	F.clear();
	V.clear();
	phi0.clear();
	W.clear();
	tmp_X.clear();
	Ri.clear();
}

void ComputeForces() {
	size_t i = 0;

	for (i = 0; i < total_points; i++) {
		F[i] = glm::vec3(0);

		//add gravity force
		if (W[i] > 0)
			F[i] += gravity;
	}
}
__device__
void IntegrateExplicitWithDamping(float deltaTime) {
	float deltaTimeMass = deltaTime;
	size_t i = 0;

	glm::vec3 Xcm = glm::vec3(0);
	glm::vec3 Vcm = glm::vec3(0);
	float sumM = 0;
	for (i = 0; i < total_points; i++) {

		V[i] *= global_dampening; //global velocity dampening !!!		
		V[i] = V[i] + (F[i] * deltaTime) * W[i];

		//calculate the center of mass's position 
		//and velocity for damping calc
		Xcm += (X[i] * mass);
		Vcm += (V[i] * mass);
		sumM += mass;
	}
	Xcm /= sumM;
	Vcm /= sumM;

	glm::mat3 I = glm::mat3(1);
	glm::vec3 L = glm::vec3(0);
	glm::vec3 w = glm::vec3(0);//angular velocity


	for (i = 0; i < total_points; i++) {
		Ri[i] = (X[i] - Xcm);

		L += glm::cross(Ri[i], mass * V[i]);

		//thanks to DevO for pointing this and these notes really helped.
		//http://www.sccg.sk/~onderik/phd/ca2010/ca10_lesson11.pdf

		glm::mat3 tmp = glm::mat3(0, -Ri[i].z, Ri[i].y,
			Ri[i].z, 0, -Ri[i].x,
			-Ri[i].y, Ri[i].x, 0);
		I += (tmp * glm::transpose(tmp)) * mass;
	}

	w = glm::inverse(I) * L;

	//apply center of mass damping
	for (i = 0; i < total_points; i++) {
		glm::vec3 delVi = Vcm + glm::cross(w, Ri[i]) - V[i];
		V[i] += kDamp * delVi;
	}

	//calculate predicted position
	for (i = 0; i < total_points; i++) {
		if (W[i] <= 0.0) {
			tmp_X[i] = X[i]; //fixed points
		}
		else {
			tmp_X[i] = X[i] + (V[i] * deltaTime);
		}
	}
}
__device__
void Integrate(float deltaTime) {
	float inv_dt = 1.0f / deltaTime;
	size_t i = 0;

	for (i = 0; i < total_points; i++) {
		V[i] = (tmp_X[i] - X[i]) * inv_dt;
		X[i] = tmp_X[i];
	}
}
__device__
void UpdateDistanceConstraint(int i) {

	DistanceConstraint c = d_constraints[i];
	glm::vec3 dir = tmp_X[c.p1] - tmp_X[c.p2];

	float len = glm::length(dir);
	if (len <= EPSILON)
		return;

	float w1 = W[c.p1];
	float w2 = W[c.p2];
	float invMass = w1 + w2;
	if (invMass <= EPSILON)
		return;

	glm::vec3 dP = (1.0f / invMass) * (len - c.rest_length) * (dir / len) * c.k_prime;
	if (w1 > 0.0)
		tmp_X[c.p1] -= dP * w1;

	if (w2 > 0.0)
		tmp_X[c.p2] += dP * w2;
}
__device__
void UpdateBendingConstraint(int index) {
	size_t i = 0;
	BendingConstraint c = b_constraints[index];

#ifdef USE_TRIANGLE_BENDING_CONSTRAINT
	//Using the paper suggested by DevO
	//http://image.diku.dk/kenny/download/kelager.niebe.ea10.pdf

	//global_k is a percentage of the global dampening constant 
	float global_k = global_dampening * 0.01f;
	glm::vec3 center = 0.3333f * (tmp_X[c.p1] + tmp_X[c.p2] + tmp_X[c.p3]);
	glm::vec3 dir_center = tmp_X[c.p3] - center;
	float dist_center = glm::length(dir_center);

	float diff = 1.0f - ((global_k + c.rest_length) / dist_center);
	glm::vec3 dir_force = dir_center * diff;
	glm::vec3 fa = c.k_prime * ((2.0f * W[c.p1]) / c.w) * dir_force;
	glm::vec3 fb = c.k_prime * ((2.0f * W[c.p2]) / c.w) * dir_force;
	glm::vec3 fc = -c.k_prime * ((4.0f * W[c.p3]) / c.w) * dir_force;

	if (W[c.p1] > 0.0) {
		tmp_X[c.p1] += fa;
	}
	if (W[c.p2] > 0.0) {
		tmp_X[c.p2] += fb;
	}
	if (W[c.p3] > 0.0) {
		tmp_X[c.p3] += fc;
	}
#else

	//Using the dihedral angle approach of the position based dynamics		
	float d = 0, phi = 0, i_d = 0;
	glm::vec3 n1 = glm::vec3(0), n2 = glm::vec3(0);

	glm::vec3 p1 = tmp_X[c.p1];
	glm::vec3 p2 = tmp_X[c.p2] - p1;
	glm::vec3 p3 = tmp_X[c.p3] - p1;
	glm::vec3 p4 = tmp_X[c.p4] - p1;

	glm::vec3 p2p3 = glm::cross(p2, p3);
	glm::vec3 p2p4 = glm::cross(p2, p4);

	float lenp2p3 = glm::length(p2p3);

	if (lenp2p3 == 0.0) { return; } //need to handle this case.

	float lenp2p4 = glm::length(p2p4);

	if (lenp2p4 == 0.0) { return; } //need to handle this case.

	n1 = glm::normalize(p2p3);
	n2 = glm::normalize(p2p4);

	d = glm::dot(n1, n2);
	phi = acos(d);

	//try to catch invalid values that will return NaN.
	// sqrt(1 - (1.0001*1.0001)) = NaN 
	// sqrt(1 - (-1.0001*-1.0001)) = NaN 
	if (d < -1.0)
		d = -1.0;
	else if (d > 1.0)
		d = 1.0; //d = clamp(d,-1.0,1.0);

	//in both case sqrt(1-d*d) will be zero and nothing will be done.
	//0?case, the triangles are facing in the opposite direction, folded together.
	if (d == -1.0) {
		phi = PI;  //acos(-1.0) == PI
		if (phi == phi0[index])
			return; //nothing to do 

	   //in this case one just need to push 
	   //vertices 1 and 2 in n1 and n2 directions, 
	   //so the constrain will do the work in second iterations.
		if (c.p1 != 0 && c.p1 != numX)
			tmp_X[c.p3] += n1 / 100.0f;

		if (c.p2 != 0 && c.p2 != numX)
			tmp_X[c.p4] += n2 / 100.0f;

		return;
	}
	if (d == 1.0) { //180?case, the triangles are planar
		phi = 0.0;  //acos(1.0) == 0.0
		if (phi == phi0[index])
			return; //nothing to do 
	}

	i_d = sqrt(1 - (d * d)) * (phi - phi0[index]);

	glm::vec3 p2n1 = glm::cross(p2, n1);
	glm::vec3 p2n2 = glm::cross(p2, n2);
	glm::vec3 p3n2 = glm::cross(p3, n2);
	glm::vec3 p4n1 = glm::cross(p4, n1);
	glm::vec3 n1p2 = -p2n1;
	glm::vec3 n2p2 = -p2n2;
	glm::vec3 n1p3 = glm::cross(n1, p3);
	glm::vec3 n2p4 = glm::cross(n2, p4);

	glm::vec3 q3 = (p2n2 + n1p2 * d) / lenp2p3;
	glm::vec3 q4 = (p2n1 + n2p2 * d) / lenp2p4;
	glm::vec3 q2 = (-(p3n2 + n1p3 * d) / lenp2p3) - ((p4n1 + n2p4 * d) / lenp2p4);

	glm::vec3 q1 = -q2 - q3 - q4;

	float q1_len2 = glm::dot(q1, q1);// glm::length(q1)*glm::length(q1);
	float q2_len2 = glm::dot(q2, q2);// glm::length(q2)*glm::length(q1);
	float q3_len2 = glm::dot(q3, q3);// glm::length(q3)*glm::length(q1);
	float q4_len2 = glm::dot(q4, q4);// glm::length(q4)*glm::length(q1); 

	float sum = W[c.p1] * (q1_len2)+
		W[c.p2] * (q2_len2)+
		W[c.p3] * (q3_len2)+
		W[c.p4] * (q4_len2);

	glm::vec3 dP1 = -((W[c.p1] * i_d) / sum) * q1;
	glm::vec3 dP2 = -((W[c.p2] * i_d) / sum) * q2;
	glm::vec3 dP3 = -((W[c.p3] * i_d) / sum) * q3;
	glm::vec3 dP4 = -((W[c.p4] * i_d) / sum) * q4;

	if (W[c.p1] > 0.0) {
		tmp_X[c.p1] += dP1 * c.k;
	}
	if (W[c.p2] > 0.0) {
		tmp_X[c.p2] += dP2 * c.k;
	}
	if (W[c.p3] > 0.0) {
		tmp_X[c.p3] += dP3 * c.k;
	}
	if (W[c.p4] > 0.0) {
		tmp_X[c.p4] += dP4 * c.k;
	}
#endif
}
//----------------------------------------------------------------------------------------------------
__device__
void GroundCollision() //DevO: 24.07.2011
{
	for (size_t i = 0; i < total_points; i++) {
		if (tmp_X[i].y < 0) //collision with ground
			tmp_X[i].y = 0;
	}
}

//----------------------------------------------------------------------------------------------------
__device__
void UpdateInternalConstraints(float deltaTime) {
	size_t i = 0;

	//printf(" UpdateInternalConstraints \n ");
	for (size_t si = 0; si < solver_iterations; ++si) {
		for (i = 0; i < d_constraints.size(); i++) {
			UpdateDistanceConstraint(i);
		}
		for (i = 0; i < b_constraints.size(); i++) {
			UpdateBendingConstraint(i);
		}
		GroundCollision();
	}
}

void OnIdle() {

	/*
		//Semi-fixed time stepping
		if ( frameTime > 0.0 )
		{
			const float deltaTime = min( frameTime, timeStep );
			StepPhysics(deltaTime );
			frameTime -= deltaTime;
		}
		*/

		//printf(" ### OnIdle %f ### \n",accumulator);
		//Fixed time stepping + rendering at different fps	
	if (accumulator >= timeStep)
	{
		StepPhysics(timeStep);
		accumulator -= timeStep;
	}

	glutPostRedisplay();
	Sleep(5); //TODO
}

void StepPhysics(float dt) {

	ComputeForces();
	IntegrateExplicitWithDamping(dt);

	// for collision constraints
	UpdateInternalConstraints(dt);

	Integrate(dt);
}

void runCuda(struct hipGraphicsResource** vbo_resource)
{
	// map OpenGL buffer object for writing from CUDA
	float4* dptr;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes,
		*vbo_resource));

	launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

void display()
{
	sdkStartTimer(&timer);

	// run CUDA kernel to generate vertex positions
	runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);

	// render from the vbo
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(4, GL_FLOAT, 0, 0);

	glEnableClientState(GL_VERTEX_ARRAY);
	glColor3f(1.0, 0.0, 0.0);
	glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
	glDisableClientState(GL_VERTEX_ARRAY);

	glutSwapBuffers();

	g_fAnim += 0.01f;

	sdkStopTimer(&timer);
	computeFPS();
}


void main(int argc, char** argv) {

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	glutInitWindowSize(width, height);
	glutCreateWindow("GLUT Cloth Demo [Position based Dynamics]");

	glutDisplayFunc(OnRender);
	glutReshapeFunc(OnReshape);
	glutIdleFunc(OnIdle);

	glutMouseFunc(OnMouseDown);
	glutMotionFunc(OnMouseMove);

	glutCloseFunc(OnShutdown);

	glewInit();
	InitGL();

	glutMainLoop();
}
